#include "hip/hip_runtime.h"
#include "utils.h"
#include "hip/hip_vector_types.h"


float3 interpolate_color(float3 oldColor, float strength, const float3& newColor) {
    float3 ret;
    ret = lerp(oldColor, newColor, strength);
    ret = clamp(ret, make_float3(0,0,0), make_float3(255,255,255));
    return ret;
}

float cosine_fallof(float val, float falloff) {
    val = powf(val, falloff);
    return (cosf(val  * (float)M_PI) + 1.0f) * 0.5f;
}


float normal_from_delta(float dx) {
    return dx / sqrtf(dx * dx + 1);
}

int2 get_coords(int x, int y, int w, int h, int width, int height) {
    const auto pixel_x = int(x / float(w) * width);
    const auto pixel_y = int(y / float(w) * height);
    return make_int2(pixel_x, pixel_y);
}