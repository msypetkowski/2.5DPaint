#include "hip/hip_runtime.h"
#include "gpu_painter.h"

#include <iostream>
#include <chrono>
#include <vector>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include ""

#include "utils.h"


const int blockSideLength = 32;


void GPUPainter::setDimensions(int w1, int h1, uchar4 *pbo) {
    args.w = w = w1;
    args.h = h = h1;

    args.pbo = buffer_pbo = pbo;

    int buf_size = w * h;

    printf("[GPU] init/resize GPU buffers (%d, %d)\n", w, h);

    if (buffer_color) { checkCudaErrors(hipFree(buffer_color)); buffer_color = nullptr; }
    if (buffer_height) { checkCudaErrors(hipFree(buffer_height)); buffer_height = nullptr; }

    checkCudaErrors(hipMalloc((void **) &buffer_height, buf_size * sizeof(float)));
    //checkCudaErrors(hipMemset(buffer_height, 0, buf_size * sizeof(float)));

    checkCudaErrors(hipMalloc((void **) &swap_buffer_height, buf_size * sizeof(float)));
    //checkCudaErrors(hipMemset(buffer_height, 0, buf_size * sizeof(float)));

    checkCudaErrors(hipMalloc((void **) &buffer_color, buf_size * sizeof(float3)));
    //checkCudaErrors(hipMemset(buffer_color, 0, buf_size * sizeof(float3)));

    args.buff_color_dptr = buffer_color;
    args.buff_height_dptr = buffer_height;
    args.swap_buff_height_dptr = swap_buffer_height;

    args.light_direction = lightDirection;

    args.blockSize = dim3(blockSideLength, blockSideLength);
    args.blocksPerGrid = dim3((w + args.blockSize.x - 1) / args.blockSize.x, (h + args.blockSize.y - 1) / args.blockSize.y);

    Painter::clear();
}

void GPUPainter::setBrushType(BrushType type) {
    using namespace std::placeholders;
    switch (type) {
        case BrushType::Default:
            paint_function = std::bind(&GPUPainter::brushBasic, this, _1, _2);
            break;
        case BrushType::Textured:
            paint_function = std::bind(&GPUPainter::brushTextured, this, _1, _2);
            break;
        case BrushType::Smooth:
            paint_function = std::bind(&GPUPainter::brushSmooth, this, _1, _2);
            break;
        default:
            throw std::runtime_error("Invalid brush type: "
                                     + std::to_string(static_cast<int>(type)));
    }
}

void GPUPainter::setTexture(const std::string &type, const unsigned char *data, int width, int height, int bytes_per_pixel) {
    int pixel_datasize = width * height * bytes_per_pixel * sizeof(unsigned char);

    unsigned char *d_color_texture = nullptr, *d_height_texture = nullptr;

    if (type == "colorFilename") {
        checkCudaErrors(hipMalloc((void **) &d_color_texture, pixel_datasize));
        checkCudaErrors(hipMemcpy(d_color_texture, data, pixel_datasize, hipMemcpyHostToDevice));
        args.ctex_height = height;
        args.ctex_width = width;
        args.ctex_bpp = bytes_per_pixel;
        args.ctex_dptr = d_color_texture;
    } else {
        checkCudaErrors(hipMalloc((void **) &d_height_texture, pixel_datasize));
        checkCudaErrors(hipMemcpy(d_height_texture, data, pixel_datasize, hipMemcpyHostToDevice));
        args.htex_height = height;
        args.htex_width = width;
        args.htex_bpp = bytes_per_pixel;
        args.htex_dptr = d_height_texture;
    }
}

void GPUPainter::doPainting(int x, int y, uchar4 *pbo) {
    auto start_time = std::chrono::steady_clock::now();
    paint_function(x, y);
    auto end_time = std::chrono::steady_clock::now();

    std::clog << "[GPU] Painting time: " <<
         (float)std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count() / (float)1e6 << " ms\n";
}

void GPUPainter::clearImage(float3 color, float height) {
    std::vector<float> fillh(w * h);
    std::fill(fillh.begin(), fillh.end(), height);

    std::vector<float3> fillcol(w * h);
    std::fill(fillcol.begin(), fillcol.end(), color);

    if (args.buff_height_dptr != nullptr) {
        checkCudaErrors(hipMemcpy(args.buff_height_dptr, fillh.data(), w * h * sizeof(float), hipMemcpyHostToDevice));
    }
    if (args.buff_height_dptr != nullptr) {
        checkCudaErrors(hipMemcpy(args.swap_buff_height_dptr, fillh.data(), w * h * sizeof(float), hipMemcpyHostToDevice));
    }
    if (args.buff_color_dptr != nullptr) {
        checkCudaErrors(hipMemcpy(args.buff_color_dptr, fillcol.data(), w * h * sizeof(float3), hipMemcpyHostToDevice));
    }

    std::clog << "[GPU] Clear image\n";
}

/**********************************************************************************************************************/
/*
 * Kernels helper functions
 */
bool __device__ inBounds(int x, int y, int w, int h) {
    return x >= 0 && x < w && y >= 0 && y < h;
}

int __device__ getBufferIndex(int x, int y, int w) { return (w - 1 - x + (y * w)); };

int __device__ getTextureByteIndex(int x, int y, int w, int h, int bpp) { return (w * bpp) * y + x * bpp; };

float __device__ sampleHeight(int x, int y, const KernelArgs &args) {
    x = clamp(x, 0, args.w - 1);
    y = clamp(y, 0, args.h - 1);
    return args.buff_height_dptr[getBufferIndex(x, y, args.w)];
}

float __device__ sampleHeight(int x, int y, int w, int h, const float *buffer) {
    x = clamp(x, 0, w - 1);
    y = clamp(y, 0, h - 1);
    return buffer[y * w + x];
}

float3 __device__ getNormalFromNeighbours(float mid, float left, float right, float top, float bottom, float bending){
    float dx = 0.0f, dy = 0.0f;

    dx += normal_from_delta(mid - right) / 2;
    dx -= normal_from_delta(mid - left) / 2;

    dy += normal_from_delta(mid - top) / 2;
    dy -= normal_from_delta(mid - bottom) / 2;

    dx *= bending;
    dy *= bending;

    dx = dx / sqrtf(dx * dx + dy * dy + 1);
    dy = dy / sqrtf(dx * dx + dy * dy + 1);

    auto ret = make_float3(dx, dy, sqrtf(fabsf(1.0f - dx * dx - dy * dy)));
    return normalize(ret);
}

float3 __device__ getNormal(int x, int y, float bending, int w, int h, const float *buffer) {
    auto mid = sampleHeight(x, y, w, h, buffer);
    auto left = sampleHeight(x - 1, y, w, h, buffer);
    auto right = sampleHeight(x + 1, y, w, h, buffer);
    auto top = sampleHeight(x, y + 1, w, h, buffer);
    auto bottom = sampleHeight(x, y - 1, w, h, buffer);
    return getNormalFromNeighbours(mid, left, right, top, bottom, bending);
}

float3 __device__ getNormal(int x, int y, float bending, const KernelArgs &args) {
    auto mid = sampleHeight(x, y, args);
    auto left = sampleHeight(x - 1, y, args);
    auto right = sampleHeight(x + 1, y, args);
    auto top = sampleHeight(x, y + 1, args);
    auto bottom = sampleHeight(x, y - 1, args);
    return getNormalFromNeighbours(mid, left, right, top, bottom, bending);
}

void GPUPainter::swapHeightBuffer() {
    float* tmp = buffer_height;
    buffer_height = swap_buffer_height;
    swap_buffer_height = tmp;

    args.buff_height_dptr = buffer_height;
    args.swap_buff_height_dptr = swap_buffer_height;
}

/**********************************************************************************************************************/
/*
 * CUDA kernels used for brush painting and display updating
 */

/*
 * Basic brush kernel
 */
__global__
void brushBasic_GPU_KERNEL(int mx, int my, const BrushSettings bs, const KernelArgs args) {

    float brush_radius = bs.size / 2.0f;

    int x = (blockIdx.x * blockDim.x) + threadIdx.x + mx - int(brush_radius);
    int y = (blockIdx.y * blockDim.y) + threadIdx.y + my - int(brush_radius);

    float radius = sqrtf((x - mx) * (x - mx) + (y - my) * (y - my));

    if (radius < brush_radius) {
        if (inBounds(x, y, args.w, args.h)) {
            // use brush

            int i = getBufferIndex(x, y, args.w);

            // paint color
            float strength = bs.pressure * cosine_fallof(radius / brush_radius, bs.falloff);
            float3 color = interpolate_color(args.buff_color_dptr[i], strength, bs.color);
            args.buff_color_dptr[i] = color;

            // paint height
            strength = bs.heightPressure * cosine_fallof(radius / brush_radius, bs.falloff);

            float result = clamp(args.buff_height_dptr[i] + strength, -1.0f, 1.0f);
            args.buff_height_dptr[i] = result;
            args.swap_buff_height_dptr[i] = result;
        }
    }
}

/*
 * Textured brush kernel
 */
__global__
void brushTextured_GPU_KERNEL(int mx, int my, const BrushSettings bs, const KernelArgs args) {

    float brush_radius = bs.size / 2.0f;

    int x = (blockIdx.x * blockDim.x) + threadIdx.x + mx - int(brush_radius);
    int y = (blockIdx.y * blockDim.y) + threadIdx.y + my - int(brush_radius);

    float radius = sqrtf((x - mx) * (x - mx) + (y - my) * (y - my));

    if (radius < brush_radius && inBounds(x, y, args.w, args.h)) {
        int i = getBufferIndex(x, y, args.w);

        float strength = bs.pressure * cosine_fallof(radius / brush_radius, bs.falloff);

        // color texture
        const auto color_coords = get_coords(x - mx + brush_radius,
                                             y - my + brush_radius,
                                             bs.size,
                                             bs.size,
                                             args.ctex_width,
                                             args.ctex_height);

        const auto pixel = getTextureByteIndex( color_coords.x,
                                                color_coords.y,
                                                args.ctex_width,
                                                args.ctex_height,
                                                args.ctex_bpp);

        args.buff_color_dptr[i] = interpolate_color(args.buff_color_dptr[i],
                                                    strength,
                                                    make_float3(args.ctex_dptr[pixel],
                                                                args.ctex_dptr[pixel + 1],
                                                                args.ctex_dptr[pixel + 2]));

        // height texture
        const auto height_coords = get_coords(  x - mx + brush_radius,
                                                y - my + brush_radius,
                                                bs.size,
                                                bs.size,
                                                args.htex_width,
                                                args.htex_height);

        const auto height = args.htex_dptr[getTextureByteIndex(height_coords.x,
                                           height_coords.y,
                                           args.htex_width,
                                           args.htex_height,
                                           args.htex_bpp)] * 0.001f;

        strength = bs.heightPressure * height * cosine_fallof(radius / brush_radius, bs.falloff);

        float result = clamp(args.buff_height_dptr[i] + strength, -1.0f, 1.0f);
        args.buff_height_dptr[i] = result;
        args.swap_buff_height_dptr[i] = result;
    }
}


/*
 * Smooth brush kernel
 */
__global__
void brushSmooth_GPU_KERNEL(int mx, int my, const BrushSettings bs, const KernelArgs args) {

    float brush_radius = bs.size / 2.0f;

    int x = (blockIdx.x * blockDim.x) + threadIdx.x + mx - int(brush_radius);
    int y = (blockIdx.y * blockDim.y) + threadIdx.y + my - int(brush_radius);

    float radius = sqrtf((x - mx) * (x - mx) + (y - my) * (y - my));

    if (radius < brush_radius) {
        if (inBounds(x, y, args.w, args.h)) {
            int i = getBufferIndex(x, y, args.w);

            float strength = cosine_fallof(radius / brush_radius, bs.falloff);

            // apply convolution filter
            auto mid = sampleHeight(x, y, args);

            auto left = sampleHeight(x - 1, y, args);
            auto right = sampleHeight(x + 1, y, args);

            auto top = sampleHeight(x, y + 1, args);
            auto bottom = sampleHeight(x, y - 1, args);

            auto topleft = sampleHeight(x - 1, y - 1, args);
            auto topright = sampleHeight(x + 1, y - 1, args);
            auto bottomleft = sampleHeight(x - 1, y + 1, args);
            auto bottomright = sampleHeight(x + 1, y + 1, args);

            float result = (mid + left + right + top + bottom + topleft + topright + bottomleft + bottomright) / 9.0f;

            args.swap_buff_height_dptr[i] = mid + strength * (result - mid);
        }
    }
}


/*
 * Kernel that updates image display
 * This kernel calculates normals based on height buffer and shades pixels properly
 * Result color is stored in pbo buffer which is rendered on the screen using OpenGL (QOpenGLWidget)
 */
__device__ __forceinline__
void updateDisplayImpl_noShm(int mx, int my, const BrushSettings bs, const KernelArgs args) {
    // printf("dupa");

    float brush_radius = bs.size / 2.0f;

    bool update_whole_display = mx == -1 && my == -1;

    int x = (blockIdx.x * blockDim.x) + threadIdx.x + (update_whole_display ? 0 : (mx - int(brush_radius)));
    int y = (blockIdx.y * blockDim.y) + threadIdx.y + (update_whole_display ? 0 : (my - int(brush_radius)));

    if (inBounds(x, y, args.w, args.h)) {
        // shading pixels
        int i = getBufferIndex(x, y, args.w);

        auto normal = getNormal(x, y, bs.normalBending, args);

        float3 color;

        if (!bs.renderNormals) {

            float3 lighting = normalize(args.light_direction);

            float shadow = fabsf(dot(lighting, normal));
            shadow = clamp(shadow, 0.0f, 1.0f);

            float specular = 1.0f - length(normal - lighting);
            specular = powf(specular, 8.0f);
            specular = clamp(specular, 0.0f, 1.0f);

            color = lerp(args.buff_color_dptr[i] * shadow, make_float3(255.0f), specular);
        } else {
            // view normals
            color.x = normal.x * 255.0 / 2 + 255.0 / 2;
            color.y = normal.y * 255.0 / 2 + 255.0 / 2;
            color.z = normal.z * 255;
        }
        color = clamp(color, make_float3(0.0f), make_float3(255.0f));
        args.pbo[i] = make_uchar4(color.x, color.y, color.z, 0);
    }
}


/*
 * Implementation of updateDisplay_GPU_KERNEL using shared memory for global memory reads optimization.
 */
__global__
void updateDisplay_GPU_KERNEL(int mx, int my, const BrushSettings bs, const KernelArgs args) {
    // Uncomment this line, and comment the rest of this function for testing
    // updateDisplayImpl_noShm(mx, my, bs, args);

    float brush_radius = bs.size / 2.0f;

    bool update_whole_display = mx == -1 && my == -1;

    // coordinates of the beginning of current block
    int bx = (blockIdx.x * blockDim.x) + (update_whole_display ? 0 : (mx - int(brush_radius)));
    int by = (blockIdx.y * blockDim.y) + (update_whole_display ? 0 : (my - int(brush_radius)));

    // coordinates inside current block
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = bx + tx;
    int y = by + ty;

    // alloc shared memory for heightmap (reduces global reads from 5 to 1 in getNormal)
    const int cache_dim = blockSideLength + 2;
    __shared__ float height_cached[cache_dim * cache_dim];
    height_cached[(ty + 1) * cache_dim + (tx + 1)] = sampleHeight(x, y, args);

    // fill borders (otherwise there would be artifacts - checker-like image)
    if (threadIdx.x == 0) {
        height_cached[(ty + 1) * cache_dim] = sampleHeight(bx - 1, y, args);
        height_cached[(ty + 1) * cache_dim + cache_dim - 1] = sampleHeight(bx - 1 + cache_dim - 1, y, args);
    }
    if (threadIdx.y == 0) {
        height_cached[tx + 1] = sampleHeight(x, by - 1, args);
        height_cached[(cache_dim - 1) * cache_dim + tx + 1] = sampleHeight(x, by - 1 + cache_dim - 1, args);
    }

    __syncthreads();

    if (inBounds(x, y, args.w, args.h)) {
        // shading pixels
        int i = getBufferIndex(x, y, args.w);

        auto normal = getNormal(tx+1, ty+1, bs.normalBending, blockSideLength + 2, blockSideLength + 2, height_cached);
        // auto normal = getNormal(x, y, bs.normalBending, args);

        float3 color;

        if (!bs.renderNormals) {

            float3 lighting = normalize(args.light_direction);

            float shadow = fabsf(dot(lighting, normal));
            shadow = clamp(shadow, 0.0f, 1.0f);

            float specular = 1.0f - length(normal - lighting);
            specular = powf(specular, 8.0f);
            specular = clamp(specular, 0.0f, 1.0f);

            color = lerp(args.buff_color_dptr[i] * shadow, make_float3(255.0f), specular);
        } else {
            // view normals
            color.x = normal.x * 255.0 / 2 + 255.0 / 2;
            color.y = normal.y * 255.0 / 2 + 255.0 / 2;
            color.z = normal.z * 255;
        }
        color = clamp(color, make_float3(0.0f), make_float3(255.0f));
        args.pbo[i] = make_uchar4(color.x, color.y, color.z, 0);
    }
}

/**********************************************************************************************************************/

/*
 * GPU painter kernels launch functions
 */


void GPUPainter::brushBasic(int mx, int my) {

    int size = int(brushSettings.size);

    args.blockSize = dim3(blockSideLength, blockSideLength);
    args.blocksPerGrid = dim3((size + args.blockSize.x - 1) / args.blockSize.x, (size + args.blockSize.y - 1) / args.blockSize.y);

    // @ TODO compute real cuda time
    brushBasic_GPU_KERNEL <<< args.blocksPerGrid, args.blockSize >>>(mx, my, brushSettings, args);
    updateDisplay_GPU_KERNEL <<< args.blocksPerGrid, args.blockSize >>>(mx, my, brushSettings, args);
    checkCudaErrors(hipDeviceSynchronize());
}


void GPUPainter::brushTextured(int mx, int my) {
    if (args.ctex_dptr == nullptr || args.htex_dptr == nullptr) {
        std::clog << "[GPU] warning: textures are not set\n";
        return;
    }

    int size = int(brushSettings.size);

    args.blockSize = dim3(blockSideLength, blockSideLength);
    args.blocksPerGrid = dim3((size + args.blockSize.x - 1) / args.blockSize.x, (size + args.blockSize.y - 1) / args.blockSize.y);

    brushTextured_GPU_KERNEL << < args.blocksPerGrid, args.blockSize >> >(mx, my, brushSettings, args);
    updateDisplay_GPU_KERNEL << < args.blocksPerGrid, args.blockSize >> >(mx, my, brushSettings, args);
    checkCudaErrors(hipDeviceSynchronize());
}


void GPUPainter::brushSmooth(int mx, int my) {
    int size = int(brushSettings.size);

    args.blockSize = dim3(blockSideLength, blockSideLength);
    args.blocksPerGrid = dim3((size + args.blockSize.x - 1) / args.blockSize.x, (size + args.blockSize.y - 1) / args.blockSize.y);

    brushSmooth_GPU_KERNEL << < args.blocksPerGrid, args.blockSize >> >(mx, my, brushSettings, args);
    updateDisplay_GPU_KERNEL << < args.blocksPerGrid, args.blockSize >> >(mx, my, brushSettings, args);
    checkCudaErrors(hipDeviceSynchronize());

    swapHeightBuffer();
}


void GPUPainter::updateWholeDisplay() {

    args.blockSize = dim3(blockSideLength, blockSideLength);
    args.blocksPerGrid = dim3((w + args.blockSize.x - 1) / args.blockSize.x, (h + args.blockSize.y - 1) / args.blockSize.y);

    updateDisplay_GPU_KERNEL << < args.blocksPerGrid, args.blockSize >> >(-1, -1, brushSettings, args);
    checkCudaErrors(hipDeviceSynchronize());
}




